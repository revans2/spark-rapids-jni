#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_utils.hpp"
#include "map_utils_debug.cuh"

#include <cudf/aggregation.hpp>
#include <cudf/binaryop.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/io/detail/tokenize_json.hpp>
#include <cudf/reduction.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/attributes.hpp>
#include <cudf/strings/find.hpp>
#include <cudf/strings/strip.hpp>
#include <cudf/unary.hpp>

#include <stdexcept>

namespace spark_rapids_jni {

std::unique_ptr<cudf::column> is_empty_or_null(
    cudf::column_view const& input, 
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) {

  auto byte_count = cudf::strings::count_bytes(cudf::strings_column_view{input}, mr); // stream not exposed yet...
  using IntScalarType = cudf::scalar_type_t<int32_t>;
  auto zero = cudf::make_numeric_scalar(cudf::data_type{cudf::type_id::INT32}, stream, mr);
  reinterpret_cast<IntScalarType *>(zero.get())->set_value(0, stream);
  zero->set_valid_async(true, stream);
  auto is_empty = cudf::binary_operation(*byte_count, *zero, cudf::binary_operator::LESS_EQUAL, cudf::data_type{cudf::type_id::BOOL8}, stream, mr);
  auto is_null = cudf::is_null(input, stream, mr);
  auto mostly_empty_or_null = cudf::binary_operation(*is_empty, *is_null, cudf::binary_operator::NULL_LOGICAL_OR, cudf::data_type{cudf::type_id::BOOL8}, stream, mr);
  is_empty.reset();
  is_null.reset();
  zero.reset();
  auto null_lit = cudf::make_string_scalar("null", stream, mr);
  auto is_lit_null = cudf::binary_operation(*null_lit, input, cudf::binary_operator::EQUAL, cudf::data_type{cudf::type_id::BOOL8}, stream, mr);
  return cudf::binary_operation(*is_lit_null, *mostly_empty_or_null, cudf::binary_operator::NULL_LOGICAL_OR, cudf::data_type{cudf::type_id::BOOL8}, stream, mr);
}

bool contains_char(
    cudf::column_view const& input,
    std::string const& needle,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) {

  cudf::string_scalar s(needle, stream, mr);
  auto has_s = cudf::strings::contains(cudf::strings_column_view(input), s);
  auto any = cudf::make_any_aggregation<cudf::reduce_aggregation>();
  auto ret = cudf::reduce(*has_s, *any, cudf::data_type{cudf::type_id::BOOL8}, mr); // no stream is supported for reduce yet
  using BoolScalarType = cudf::scalar_type_t<bool>;
  return ret->is_valid(stream) && reinterpret_cast<BoolScalarType *>(ret.get())->value(stream);
}

std::pair<std::unique_ptr<cudf::column>, std::unique_ptr<cudf::column>> clean(
    cudf::column_view const& input,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) {
  auto const input_scv  = cudf::strings_column_view{input};
  auto stripped = cudf::strings::strip(input_scv, cudf::strings::side_type::BOTH, cudf::string_scalar(""), stream, mr);
  auto is_n_or_e = is_empty_or_null(*stripped, stream, mr);
  auto empty_row = cudf::make_string_scalar("{}", stream, mr);
  auto cleaned = cudf::copy_if_else(*empty_row, *stripped, *is_n_or_e, stream, mr);
  stripped.reset();
  empty_row.reset();
  if (contains_char(*cleaned, "\n", stream, mr)) {
    throw std::logic_error("line separator is not currently supported in a JSON string");
  }
  if (contains_char(*cleaned, "\r", stream, mr)) {
    throw std::logic_error("carriage return is not currently supported in a JSON string");
  }



  // TODO probably want to have/use a data source instead of a concat buffer.

  /*
  auto const d_strings  = cudf::column_device_view::create(input, stream);
  auto const chars_size = input_scv.chars_size(stream);
  auto const output_size =
    static_cast<int64_t>(chars_size) +
    static_cast<int64_t>(input.size() - 1) +        // append `\n` character between input rows
    static_cast<int64_t>(input.null_count()) * 2l;  // replace null with "{}" (we probably want to deal with empty strings too)
  // TODO: This assertion eventually needs to be removed.
  // See https://github.com/NVIDIA/spark-rapids-jni/issues/1707
  CUDF_EXPECTS(output_size <= static_cast<int64_t>(std::numeric_limits<cudf::size_type>::max()),
               "The input json column is too large and causes overflow.");

  auto const joined_input = cudf::strings::detail::join_strings(
    input_scv,
    cudf::string_scalar("\n"),   // append `,` character between the input rows
    cudf::string_scalar("{}"),  // replacement for null rows
    stream,
    mr);
  auto const joined_input_scv        = cudf::strings_column_view{*joined_input};
  auto const joined_input_size_bytes = joined_input_scv.chars_size(stream);
  // TODO: This assertion requires a stream synchronization, may want to remove at some point.
  // See https://github.com/NVIDIA/spark-rapids-jni/issues/1707
  CUDF_EXPECTS(joined_input_size_bytes + 2 == output_size, "Incorrect output size computation.");

  // We want to concatenate 3 strings: "[" + joined_input + "]".
  // For efficiency, let's use memcpy instead of `cudf::strings::detail::concatenate`.
  auto output = rmm::device_uvector<char>(joined_input_size_bytes + 2, stream);
  CUDF_CUDA_TRY(hipMemsetAsync(output.data(), static_cast<int>('['), 1, stream.value()));
  CUDF_CUDA_TRY(hipMemcpyAsync(output.data() + 1,
                                joined_input_scv.chars_begin(stream),
                                joined_input_size_bytes,
                                hipMemcpyDefault,
                                stream.value()));
  CUDF_CUDA_TRY(hipMemsetAsync(
    output.data() + joined_input_size_bytes + 1, static_cast<int>(']'), 1, stream.value()));

#ifdef DEBUG_FROM_JSON
  print_debug<char, char>(output, "Processed json string", "", stream);
#endif
  return output;

  if (input.data
  */
  throw std::runtime_error("NOT DONE YET");
}

std::unique_ptr<cudf::column> tokenize_json(
  cudf::column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr) {

  CUDF_EXPECTS(input.type().id() == cudf::type_id::STRING, "Invalid input format");

  if (input.is_empty()) {
    auto tok_out = cudf::make_empty_column(cudf::type_id::INT8);
    auto offset_out = cudf::make_empty_column(cudf::type_id::UINT32);
    std::vector<std::unique_ptr<cudf::column>> tok_off_children;
    tok_off_children.push_back(std::move(tok_out));
    tok_off_children.push_back(std::move(offset_out));
    auto tok_off_out = cudf::make_structs_column(0, std::move(tok_off_children), 0, rmm::device_buffer{}, stream, mr);
    auto empty_offsets = cudf::make_empty_column(cudf::type_id::INT32);
    auto tokens_out = cudf::make_lists_column(0, std::move(empty_offsets), std::move(tok_off_out), 0, rmm::device_buffer{}, stream, mr);
    auto buffer_out = cudf::make_empty_column(cudf::type_id::STRING);
    std::vector<std::unique_ptr<cudf::column>> children;
    children.push_back(std::move(buffer_out));
    children.push_back(std::move(tokens_out));
    return cudf::make_structs_column(0, std::move(children), 0, rmm::device_buffer{}, stream, mr);
  }

  auto [cleaned, was_empty] = clean(input, stream, mr);

  // TODO we probably want a JSON options to pass in at some point. For now we are
  // just going to hard code thigns...

  // First off we need to get all of the data into a single buffer.  In the future
  // This will use \0 nul as the separator, but for now we are going to use \n
  // and check that it is not in there...

  throw std::runtime_error("NOT IMPLEMENTED YET");
}

}  // namespace spark_rapids_jni
